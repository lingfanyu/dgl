#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2019 by Contributors
 * \file kernel/cuda/utils.cu
 * \brief Utility function implementations on CUDA
 */
#include "../../runtime/cuda/cuda_common.h"
#include "../utils.h"

namespace dgl {
namespace kernel {
namespace utils {

template <typename DType>
__global__ void _FillKernel(DType* ptr, size_t length, DType val) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    ptr[tx] = val;
    tx += stride_x;
  }
}

template <int XPU, typename DType>
void Fill(const DLContext& ctx, DType* ptr, size_t length, DType val) {
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  int nt = utils::FindNumThreads(length, 1024);
  int nb = (length + nt - 1) / nt;
  _FillKernel<<<nb, nt, 0, thr_entry->stream>>>(ptr, length, val);
}

template void Fill<kDLGPU, float>(const DLContext& ctx, float* ptr, size_t length, float val);
template void Fill<kDLGPU, double>(const DLContext& ctx, double* ptr, size_t length, double val);

template <typename Idx, typename DType>
__global__ void _Take1DKernel(DType* ptr, const DType* src, const Idx* indices, size_t length) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    ptr[tx] = src[indices[tx]];
    tx += stride_x;
  }
}

template <int XPU, typename Idx, typename DType>
void Take1D(const DLContext& ctx, DType* ptr, const DType* src, const Idx* indices, size_t length) {
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  int nt = utils::FindNumThreads(length, 1024);
  int nb = (length + nt - 1) / nt;
  _Take1DKernel<<<nb, nt, 0, thr_entry->stream>>>(ptr, src, indices, length);
}

template void Take1D<kDLGPU, int32_t, float>(const DLContext&, float*, const float*, const int32_t*, size_t);
template void Take1D<kDLGPU, int32_t, double>(const DLContext&, double*, const double*, const int32_t*, size_t);

}  // namespace utils
}  // namespace kernel
}  // namespace dgl
