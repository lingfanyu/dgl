#include "./cusparse_common.cuh"

namespace dgl {
namespace kernel {
namespace cuda {

template <typename DType>
hipsparseStatus_t Xcsrmm2(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const DType* alpha, const hipsparseMatDescr_t descrA,
    const DType* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const DType* B, int ldb, const DType* beta, DType* C, int ldc) {
  LOG(INFO) << "Not supported dtype";
  return HIPSPARSE_STATUS_EXECUTION_FAILED;
}

template <>
hipsparseStatus_t Xcsrmm2<float>(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const float* alpha, const hipsparseMatDescr_t descrA,
    const float* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const float* B, int ldb, const float* beta, float* C, int ldc) {
  return hipsparseScsrmm2(handle, transA, transB, m, n, k, nnz,
      alpha, descrA, csrValA, csrRowPtrA, csrColIndA,
      B, ldb, beta, C, ldc);
}

template <>
hipsparseStatus_t Xcsrmm2<double>(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const double* alpha, const hipsparseMatDescr_t descrA,
    const double* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const double* B, int ldb, const double* beta, double* C, int ldc) {
  return hipsparseDcsrmm2(handle, transA, transB, m, n, k, nnz,
      alpha, descrA, csrValA, csrRowPtrA, csrColIndA,
      B, ldb, beta, C, ldc);
}

template <typename DType>
hipblasStatus_t Xgeam(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const DType* alpha, const DType* A, int lda,
    const DType* beta, const DType* B, int ldb,
    DType* C, int ldc) {
  LOG(INFO) << "Not supported dtype";
  return HIPBLAS_STATUS_EXECUTION_FAILED;
}

template <>
hipblasStatus_t Xgeam<float>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const float* alpha, const float* A, int lda,
    const float* beta, const float* B, int ldb,
    float* C, int ldc) {
  return hipblasSgeam(handle, transa, transb, m, n, alpha, A, lda,
      beta, B, ldb, C, ldc);
}

template <>
hipblasStatus_t Xgeam<double>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const double* alpha, const double* A, int lda,
    const double* beta, const double* B, int ldb,
    double* C, int ldc) {
  return hipblasDgeam(handle, transa, transb, m, n, alpha, A, lda,
      beta, B, ldb, C, ldc);
}

} // namespace cuda
} // namespace kernel
} // namespace dgl
